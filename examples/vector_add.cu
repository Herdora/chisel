// vector_add.cu
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" void launch_vector_add(const float* A, const float* B, float* C, int N);

__global__ void vector_add_kernel(const float* A, const float* B, float* C, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

extern "C" void launch_vector_add(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    vector_add_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
