#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simple_kernel(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 1000000;
    const int size = N * sizeof(float);
    
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    // Initialize with some data
    float *h_a = new float[N];
    float *h_b = new float[N];
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    // Launch kernel multiple times
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    for (int i = 0; i < 10; i++) {
        simple_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    }
    
    hipDeviceSynchronize();
    
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    
    std::cout << "Kernel execution completed!" << std::endl;
    return 0;
}